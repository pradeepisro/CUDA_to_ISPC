#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Computation of eigenvalues of symmetric, tridiagonal matrix using
 * bisection.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <assert.h>
#include <string>
#include <algorithm>
#include <fstream>
#include <vector>

// includes, project
#include "config.h"
#include "structs.h"
#include "util.h"

#include "bisect_large.cuh"
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char **argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
    bool bQAResults = false;

    printf("Starting eigenvalues\n");

    bQAResults = runTest(argc, argv);
    printf("Test %s\n", bQAResults ? "Succeeded!" : "Failed!");

    exit(bQAResults ? EXIT_SUCCESS : EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize the input data to the algorithm
//! @param input  handles to the input data
//! @param exec_path  path where executable is run (argv[0])
//! @param mat_size  size of the matrix
//! @param user_defined  1 if the matrix size has been requested by the user,
//!                      0 if the default size
////////////////////////////////////////////////////////////////////////////////
void
initInputData(InputData &input, char *exec_path,
              const unsigned int mat_size, const unsigned int user_defined)
{
    // allocate memory
    input.a = (float *) malloc(sizeof(float) * mat_size);
    input.b = (float *) malloc(sizeof(float) * mat_size);

    if (user_defined == 1)
    {

        // initialize diagonal and superdiagonal entries with random values
        srand(278217421);

        // srand( clock());
        for (unsigned int i = 0; i < mat_size; ++i)
        {
            input.a[i] = (float)(2.0 * (((double)rand()
                                         / (double) RAND_MAX) - 0.5));
            input.b[i] = (float)(2.0 * (((double)rand()
                                         / (double) RAND_MAX) - 0.5));
        }

        // the first element of s is used as padding on the device (thus the
        // whole vector is copied to the device but the kernels are launched
        // with (s+1) as start address
        input.b[0] = 0.0f;
    }
    else
    {

        // read default matrix
        unsigned int input_data_size = mat_size;
        std::vector<float> buffer;
        buffer.reserve(input_data_size);
        std::fstream file("./data/diagonal.dat");
        for(size_t i = 0, temp = 0; i < mat_size; i++){
            file >> temp;
            buffer.push_back(temp);
        }
        input.a = new float[input_data_size];
        std::copy(input.a, input.a + input_data_size, buffer.begin());
        file.close();

        file.open("./data/superdiagonal.dat");
        for(size_t i = 0, temp = 0; i < mat_size; i++){
            file >> temp;
            buffer[i] = temp;
        }
        input.b = new float[input_data_size];
        std::copy(input.b, input.b + input_data_size, buffer.begin());
    }

    // allocate device memory for input
    hipMalloc((void **) &(input.g_b_raw), sizeof(float) * mat_size);
    hipMalloc((void **) &(input.g_a)    , sizeof(float) * mat_size);

    // copy data to device
    hipMemcpy(input.g_b_raw, input.b, sizeof(float) * mat_size, hipMemcpyHostToDevice);
    hipMemcpy(input.g_a    , input.a, sizeof(float) * mat_size, hipMemcpyHostToDevice);

    input.g_b = input.g_b_raw + 1;
}

////////////////////////////////////////////////////////////////////////////////
//! Clean up input data, in particular allocated memory
//! @param input  handles to the input data
////////////////////////////////////////////////////////////////////////////////
void
cleanupInputData(InputData &input)
{

    freePtr(input.a);
    freePtr(input.b);

    hipFree(input.g_a);
    input.g_a = NULL;
    hipFree(input.g_b_raw);
    input.g_b_raw = NULL;
    input.g_b = NULL;
}

////////////////////////////////////////////////////////////////////////////////
//! Check if a specific matrix size has to be used
//! @param argc  number of command line arguments (from main(argc, argv)
//! @param argv  pointers to command line arguments (from main(argc, argv)
//! @param matrix_size  size of matrix, updated if specific size specified on
//!                     command line
////////////////////////////////////////////////////////////////////////////////
void
getMatrixSize(int argc, char **argv,
              unsigned int &mat_size, unsigned int &user_defined)
{
/*     int temp = -1;

    if (checkCmdLineFlag(argc, (const char **)argv, "matrix-size"))
    {
        temp = getCmdLineArgumentInt(argc, (const char **) argv, "matrix-size");
    }

    if (temp > 0)
    {

        mat_size = (unsigned int) temp;
        // data type short is used in the kernel
        assert(mat_size < (1 << 16));

        user_defined = 1;
    } */

    printf("Matrix size: %i x %i\n", mat_size, mat_size);
}

////////////////////////////////////////////////////////////////////////////////
//! Check if a specific precision of the eigenvalue has to be obtained
//! @param argc  number of command line arguments (from main(argc, argv)
//! @param argv  pointers to command line arguments (from main(argc, argv)
//! @param iters_timing  numbers of iterations for timing, updated if a
//!                      specific number is specified on the command line
////////////////////////////////////////////////////////////////////////////////
void
getPrecision(int argc, char **argv, float &precision)
{

/*     float temp = -1.0f;

    if (checkCmdLineFlag(argc, (const char **)argv, "precision"))
    {
        temp = getCmdLineArgumentFloat(argc, (const char **) argv, "precision");
    }

    if (temp > 0.0f)
    {
        precision = temp;
    } */

    printf("Precision: %f\n", precision);
}

////////////////////////////////////////////////////////////////////////////////
//! Check if a particular number of iterations for timings has to be used
//! @param argc  number of command line arguments (from main(argc, argv)
//! @param argv  pointers to command line arguments (from main(argc, argv)
//! @param  iters_timing  number of timing iterations, updated if user
//!                       specific value
////////////////////////////////////////////////////////////////////////////////
void
getItersTiming(int argc, char **argv, unsigned int &iters_timing)
{

    /* int temp = -1;

    if (checkCmdLineFlag(argc, (const char **)argv, "iters-timing"))
    {
        temp = getCmdLineArgumentInt(argc, (const char **) argv, "iters-timing");
    }

    if (temp > 0)
    {
        iters_timing = temp;
    } */

    printf("Iterations to be timed: %i\n", iters_timing);
}

////////////////////////////////////////////////////////////////////////////////
//! Check if a particular filename has to be used for the file where the result
//! is stored
//! @param argc  number of command line arguments (from main(argc, argv)
//! @param argv  pointers to command line arguments (from main(argc, argv)
//! @param  filename  filename of result file, updated if user specified
//!                   filename
////////////////////////////////////////////////////////////////////////////////
void
getResultFilename(int argc, char **argv, const char *&filename)
{

    /* char *temp = NULL;
    getCmdLineArgumentString(argc, (const char **) argv, "filename-result",
                             &temp);

    if (NULL != temp)
    {

        filename = (char *) malloc(sizeof(char) * strlen(temp));
        strcpy(filename, temp);

        free(temp);
    } */

    printf("Result filename: '%s'\n", filename);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool
runTest(int argc, char **argv)
{
    bool bCompareResult = false;

    // default
    unsigned int mat_size = 2048;
    // flag if the matrix size is due to explicit user request
    unsigned int user_defined = 0;
    // desired precision of eigenvalues
    float  precision = 0.00001f;
    unsigned int iters_timing = 100;
    const char* result_file = "eigenvalues.dat";

    // check if there is a command line request for the matrix size
    getMatrixSize(argc, argv, mat_size, user_defined);

    // check if user requested specific precision
    getPrecision(argc, argv, precision);

    // check if user requested specific number of iterations for timing
    getItersTiming(argc, argv, iters_timing);

    // file name for result file
    getResultFilename(argc, argv, result_file);

    // set up input
    InputData input;
    initInputData(input, argv[0], mat_size, user_defined);

    // compute Gerschgorin interval
    float lg = FLT_MAX;
    float ug = -FLT_MAX;
        ResultDataLarge  result;
        initResultDataLargeMatrix(result, mat_size);

        // run the kernel
        computeEigenvaluesLargeMatrix(input, result, mat_size,
                                      precision, lg, ug,
                                      iters_timing);

        // get the result from the device and do some sanity checks
        // save the result if user specified matrix size
        bCompareResult = processResultDataLargeMatrix(input, result, mat_size, result_file,
                                                      user_defined, argv[0]);

        // cleanup
        cleanupResultDataLargeMatrix(result);
    // }

    cleanupInputData(input);
    // hipDeviceReset();

    return bCompareResult;
}
