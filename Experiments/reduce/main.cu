#include <algorithm>
#include <iostream>
#include <numeric>
#include <vector>

#include "cuda_utils.cuh"
#include "reduce_cuda.cuh"
#include "reduce_ispc.h"

template <typename T> T executeReference(std::vector<T> &v) {
    return std::accumulate(v.begin(), v.end(), 0);
}

int executeCUDA(std::vector<int> &v) {
    std::vector<int> u(v.begin(), v.end());
    int *d_a = nullptr, *d_partial_sum;
    size_t N = u.size();
    size_t nthreads = N > 1024 ? 1024 : N;
    size_t nblocks = N > 1024 ? (N - 1) / 1024 + 1 : 1;
    cudaCheck(hipMalloc((void **)&d_a, N * sizeof(int)));
    cudaCheck(hipMalloc((void **)&d_partial_sum, nblocks * sizeof(int)));
    cudaCheck(
        hipMemcpy(d_a, u.data(), N * sizeof(int), hipMemcpyHostToDevice));
    cudaCheckLaunch(reduce, nblocks, nthreads, d_a, d_partial_sum, nthreads);
    hipDeviceSynchronize();
    int partial_sum[nblocks];
    cudaCheck(hipMemcpy(partial_sum, d_partial_sum, sizeof(int) * nblocks, hipMemcpyDeviceToHost));
    int sum = 0;
    for(int i = 0; i < nblocks; i++)
        sum += partial_sum[i];
    return sum;
}

int executeISPC(std::vector<int> &v) {
    std::vector<int> u(v.begin(), v.end());
    size_t N = u.size();
    int nthreads = N > 1024 ? 1024 : N;
    int nblocks = N > 1024 ? (N - 1) / 1024 + 1 : 1;
    std::cout << nblocks << ", " << nthreads << '\n';
    ispc::Dim3 grid = {nblocks, 1, 1};
    ispc::Dim3 block = {nthreads, 1, 1};
    int *partial_sum = new int[nblocks];
    ispc::reduce(grid, block, u.data(), partial_sum, nthreads);
    int sum = 0;
    for(int i = 0; i < nblocks; i++)
        sum += partial_sum[i];
    return sum;
}

int main(int argc, char *argv[]) {
    size_t N = 1 << 14;
    std::vector<int> v(N);
    std::iota(v.begin(), v.end(), 0);
    std::cout << executeReference(v) << '\n';
    std::cout << executeCUDA(v) << '\n';
    std::cout << executeISPC(v) << '\n';
    return 0;
}