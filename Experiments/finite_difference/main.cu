#include "hip/hip_runtime.h"
// #include "finite_difference_cuda.cuh"
#include "finite_difference_ispc.h"
#include <cuda_utils.cuh>
#include <iostream>

// shared memory tiles will be m*-by-*Pencils
// sPencils is used when each thread calculates the derivative at one point
// lPencils is used for coalescing in y and z where each thread has to
//     calculate the derivative at mutiple points

dim3 grid[3][2], block[3][2];

ispc::ConstantMemory cm;

float fx = 1.0f, fy = 1.0f, fz = 1.0f;
const int mx = 64, my = 64, mz = 64;

const int sPencils = 4;
const int lPencils = 32;

__constant__ float c_ax, c_bx, c_cx, c_dx;
__constant__ float c_ay, c_by, c_cy, c_dy;
__constant__ float c_az, c_bz, c_cz, c_dz;

__global__ void derivative_x(float *f, float *df) {
    __shared__ float s_f[sPencils][mx + 8]; // 4-wide halo

    //   if(blockIdx.x == 0 && threadIdx.x == 0){
    //       printf("%f, %f, %f, %f\n", c_ax, c_bx, c_cx, c_dx);
    //   }

    int i = threadIdx.x;
    int j = blockIdx.x * blockDim.y + threadIdx.y;
    int k = blockIdx.y;
    int si = i + 4;       // local i for shared memory access + halo offset
    int sj = threadIdx.y; // local j for shared memory access

    int globalIdx = k * mx * my + j * mx + i;

    s_f[sj][si] = f[globalIdx];

    __syncthreads();

    // fill in periodic images in shared memory array
    if (i < 4) {
        s_f[sj][si - 4] = s_f[sj][si + mx - 5];
        s_f[sj][si + mx] = s_f[sj][si + 1];
    }

    __syncthreads();

    df[globalIdx] = (c_ax * (s_f[sj][si + 1] - s_f[sj][si - 1]) +
                     c_bx * (s_f[sj][si + 2] - s_f[sj][si - 2]) +
                     c_cx * (s_f[sj][si + 3] - s_f[sj][si - 3]) +
                     c_dx * (s_f[sj][si + 4] - s_f[sj][si - 4]));
}

__global__ void derivative_x_lPencils(float *f, float *df) {
    __shared__ float s_f[lPencils][mx + 8]; // 4-wide halo

    int i = threadIdx.x;
    int jBase = blockIdx.x * lPencils;
    int k = blockIdx.y;
    int si = i + 4; // local i for shared memory access + halo offset

    for (int sj = threadIdx.y; sj < lPencils; sj += blockDim.y) {
        int globalIdx = k * mx * my + (jBase + sj) * mx + i;
        s_f[sj][si] = f[globalIdx];
    }

    __syncthreads();

    // fill in periodic images in shared memory array
    if (i < 4) {
        for (int sj = threadIdx.y; sj < lPencils; sj += blockDim.y) {
            s_f[sj][si - 4] = s_f[sj][si + mx - 5];
            s_f[sj][si + mx] = s_f[sj][si + 1];
        }
    }

    __syncthreads();

    for (int sj = threadIdx.y; sj < lPencils; sj += blockDim.y) {
        int globalIdx = k * mx * my + (jBase + sj) * mx + i;
        df[globalIdx] = (c_ax * (s_f[sj][si + 1] - s_f[sj][si - 1]) +
                         c_bx * (s_f[sj][si + 2] - s_f[sj][si - 2]) +
                         c_cx * (s_f[sj][si + 3] - s_f[sj][si - 3]) +
                         c_dx * (s_f[sj][si + 4] - s_f[sj][si - 4]));
    }
}

__global__ void derivative_y(float *f, float *df) {
    __shared__ float s_f[my + 8][sPencils];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = threadIdx.y;
    int k = blockIdx.y;
    int si = threadIdx.x;
    int sj = j + 4;

    int globalIdx = k * mx * my + j * mx + i;

    s_f[sj][si] = f[globalIdx];

    __syncthreads();

    if (j < 4) {
        s_f[sj - 4][si] = s_f[sj + my - 5][si];
        s_f[sj + my][si] = s_f[sj + 1][si];
    }

    __syncthreads();

    df[globalIdx] = (c_ay * (s_f[sj + 1][si] - s_f[sj - 1][si]) +
                     c_by * (s_f[sj + 2][si] - s_f[sj - 2][si]) +
                     c_cy * (s_f[sj + 3][si] - s_f[sj - 3][si]) +
                     c_dy * (s_f[sj + 4][si] - s_f[sj - 4][si]));
}

__global__ void derivative_y_lPencils(float *f, float *df) {
    __shared__ float s_f[my + 8][lPencils];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y;
    int si = threadIdx.x;

    for (int j = threadIdx.y; j < my; j += blockDim.y) {
        int globalIdx = k * mx * my + j * mx + i;
        int sj = j + 4;
        s_f[sj][si] = f[globalIdx];
    }

    __syncthreads();

    int sj = threadIdx.y + 4;
    if (sj < 8) {
        s_f[sj - 4][si] = s_f[sj + my - 5][si];
        s_f[sj + my][si] = s_f[sj + 1][si];
    }

    __syncthreads();

    for (int j = threadIdx.y; j < my; j += blockDim.y) {
        int globalIdx = k * mx * my + j * mx + i;
        int sj = j + 4;
        df[globalIdx] = (c_ay * (s_f[sj + 1][si] - s_f[sj - 1][si]) +
                         c_by * (s_f[sj + 2][si] - s_f[sj - 2][si]) +
                         c_cy * (s_f[sj + 3][si] - s_f[sj - 3][si]) +
                         c_dy * (s_f[sj + 4][si] - s_f[sj - 4][si]));
    }
}

__global__ void derivative_z(float *f, float *df) {
    __shared__ float s_f[mz + 8][sPencils];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y;
    int k = threadIdx.y;
    int si = threadIdx.x;
    int sk = k + 4; // halo offset

    int globalIdx = k * mx * my + j * mx + i;

    s_f[sk][si] = f[globalIdx];

    __syncthreads();

    if (k < 4) {
        s_f[sk - 4][si] = s_f[sk + mz - 5][si];
        s_f[sk + mz][si] = s_f[sk + 1][si];
    }

    __syncthreads();

    df[globalIdx] = (c_az * (s_f[sk + 1][si] - s_f[sk - 1][si]) +
                     c_bz * (s_f[sk + 2][si] - s_f[sk - 2][si]) +
                     c_cz * (s_f[sk + 3][si] - s_f[sk - 3][si]) +
                     c_dz * (s_f[sk + 4][si] - s_f[sk - 4][si]));
}

__global__ void derivative_z_lPencils(float *f, float *df) {
    __shared__ float s_f[mz + 8][lPencils];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y;
    int si = threadIdx.x;

    for (int k = threadIdx.y; k < mz; k += blockDim.y) {
        int globalIdx = k * mx * my + j * mx + i;
        int sk = k + 4;
        s_f[sk][si] = f[globalIdx];
    }

    __syncthreads();

    int k = threadIdx.y + 4;
    if (k < 8) {
        s_f[k - 4][si] = s_f[k + mz - 5][si];
        s_f[k + mz][si] = s_f[k + 1][si];
    }

    __syncthreads();

    for (int k = threadIdx.y; k < mz; k += blockDim.y) {
        int globalIdx = k * mx * my + j * mx + i;
        int sk = k + 4;
        df[globalIdx] = (c_az * (s_f[sk + 1][si] - s_f[sk - 1][si]) +
                         c_bz * (s_f[sk + 2][si] - s_f[sk - 2][si]) +
                         c_cz * (s_f[sk + 3][si] - s_f[sk - 3][si]) +
                         c_dz * (s_f[sk + 4][si] - s_f[sk - 4][si]));
    }
}

// host routine to set constant data
void setDerivativeParameters() {
    // check to make sure dimensions are integral multiples of sPencils
    if ((mx % sPencils != 0) || (my % sPencils != 0) || (mz % sPencils != 0)) {
        printf("'mx', 'my', and 'mz' must be integral multiples of sPencils\n");
        exit(1);
    }

    if ((mx % lPencils != 0) || (my % lPencils != 0)) {
        printf("'mx' and 'my' must be multiples of lPencils\n");
        exit(1);
    }

    // stencil weights (for unit length problem)
    float dsinv = mx - 1.f;

    float ax = 4.f / 5.f * dsinv;
    float bx = -1.f / 5.f * dsinv;
    float cx = 4.f / 105.f * dsinv;
    float dx = -1.f / 280.f * dsinv;
    cm.c_ax = ax, cm.c_bx = bx, cm.c_cx = cx, cm.c_dx = dx;
    cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_ax), &ax, sizeof(float), 0,
                                 hipMemcpyHostToDevice));
    cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_bx), &bx, sizeof(float), 0,
                                 hipMemcpyHostToDevice));
    cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_cx), &cx, sizeof(float), 0,
                                 hipMemcpyHostToDevice));
    cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_dx), &dx, sizeof(float), 0,
                                 hipMemcpyHostToDevice));

    dsinv = my - 1.f;

    float ay = 4.f / 5.f * dsinv;
    float by = -1.f / 5.f * dsinv;
    float cy = 4.f / 105.f * dsinv;
    float dy = -1.f / 280.f * dsinv;
    cm.c_ay = ay, cm.c_by = by, cm.c_cy = cy, cm.c_dy = dy;
    cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_ay), &ay, sizeof(float), 0,
                                 hipMemcpyHostToDevice));
    cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_by), &by, sizeof(float), 0,
                                 hipMemcpyHostToDevice));
    cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_cy), &cy, sizeof(float), 0,
                                 hipMemcpyHostToDevice));
    cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_dy), &dy, sizeof(float), 0,
                                 hipMemcpyHostToDevice));

    dsinv = mz - 1.f;

    float az = 4.f / 5.f * dsinv;
    float bz = -1.f / 5.f * dsinv;
    float cz = 4.f / 105.f * dsinv;
    float dz = -1.f / 280.f * dsinv;
    cm.c_az = az, cm.c_bz = bz, cm.c_cz = cz, cm.c_dz = dz;
    cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_az), &az, sizeof(float), 0,
                                 hipMemcpyHostToDevice));
    cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_bz), &bz, sizeof(float), 0,
                                 hipMemcpyHostToDevice));
    cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_cz), &cz, sizeof(float), 0,
                                 hipMemcpyHostToDevice));
    cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_dz), &dz, sizeof(float), 0,
                                 hipMemcpyHostToDevice));

    // Execution configurations for small and large pencil tiles

    grid[0][0] = dim3(my / sPencils, mz, 1);
    block[0][0] = dim3(mx, sPencils, 1);

    grid[0][1] = dim3(my / lPencils, mz, 1);
    block[0][1] = dim3(mx, sPencils, 1);

    grid[1][0] = dim3(mx / sPencils, mz, 1);
    block[1][0] = dim3(sPencils, my, 1);

    grid[1][1] = dim3(mx / lPencils, mz, 1);
    // we want to use the same number of threads as above,
    // so when we use lPencils instead of sPencils in one
    // dimension, we multiply the other by sPencils/lPencils
    block[1][1] = dim3(lPencils, my * sPencils / lPencils, 1);

    grid[2][0] = dim3(mx / sPencils, my, 1);
    block[2][0] = dim3(sPencils, mz, 1);

    grid[2][1] = dim3(mx / lPencils, my, 1);
    block[2][1] = dim3(lPencils, mz * sPencils / lPencils, 1);
}

void initInput(float *f, int dim) {
    const float twopi = 8.f * (float)atan(1.0);

    for (int k = 0; k < mz; k++) {
        for (int j = 0; j < my; j++) {
            for (int i = 0; i < mx; i++) {
                switch (dim) {
                case 0:
                    f[k * mx * my + j * mx + i] =
                        cos(fx * twopi * (i - 1.f) / (mx - 1.f));
                    break;
                case 1:
                    f[k * mx * my + j * mx + i] =
                        cos(fy * twopi * (j - 1.f) / (my - 1.f));
                    break;
                case 2:
                    f[k * mx * my + j * mx + i] =
                        cos(fz * twopi * (k - 1.f) / (mz - 1.f));
                    break;
                }
            }
        }
    }
}

void initSol(float *sol, int dim) {
    const float twopi = 8.f * (float)atan(1.0);

    for (int k = 0; k < mz; k++) {
        for (int j = 0; j < my; j++) {
            for (int i = 0; i < mx; i++) {
                switch (dim) {
                case 0:
                    sol[k * mx * my + j * mx + i] =
                        -fx * twopi * sin(fx * twopi * (i - 1.f) / (mx - 1.f));
                    break;
                case 1:
                    sol[k * mx * my + j * mx + i] =
                        -fy * twopi * sin(fy * twopi * (j - 1.f) / (my - 1.f));
                    break;
                case 2:
                    sol[k * mx * my + j * mx + i] =
                        -fz * twopi * sin(fz * twopi * (k - 1.f) / (mz - 1.f));
                    break;
                }
            }
        }
    }
}

void checkResults(double &error, double &maxError, float *sol, float *df) {
    // error = sqrt(sum((sol-df)**2)/(mx*my*mz))
    // maxError = maxval(abs(sol-df))
    maxError = 0;
    error = 0;
    size_t mismatch_count = 0;
    size_t total_count = 0;
    for (int k = 0; k < mz; k++) {
        for (int j = 0; j < my; j++) {
            for (int i = 0; i < mx; i++) {
                float s = sol[k * mx * my + j * mx + i];
                float f = df[k * mx * my + j * mx + i];
                if ((int)(s * 1000) != (int)(f * 1000)) {
                    mismatch_count++;
                    // printf("%d %d %d: %f %f\n", i, j, k, s, f);
                }
                total_count++;
                error += (s - f) * (s - f);
                if (fabs(s - f) > maxError)
                    maxError = fabs(s - f);
            }
        }
    }
    std::cout << "total count: " << total_count << "\n";
    std::cout << "MisMathed count: " << mismatch_count << "\n";
    error = sqrt(error / (mx * my * mz));
}

// void printResults(float *cuda_f, float *ispc_f){

// }

// Run the kernels for a given dimension. One for sPencils, one for lPencils
void runTest(int dimension) {
    void (*fpDeriv[2])(float *, float *);
    void (*ispcDeriv[2])(const struct ispc::ConstantMemory &, float *,
                         struct ispc::gridDim &, struct ispc::blockDim &,
                         float *, float *, const int32_t, const int32_t,
                         const int32_t);
    float *ispc_sm_ptr[2];
    const int pencil[2] = {sPencils, lPencils};
    switch (dimension) {
    case 0:
        fpDeriv[0] = derivative_x;
        fpDeriv[1] = derivative_x_lPencils;
        ispcDeriv[0] = ispc::derivative_x_ispc;
        ispcDeriv[1] = ispc::derivative_x_lPencils_ispc;
        ispc_sm_ptr[0] = new float[sPencils * (mx + 8)];
        ispc_sm_ptr[1] = new float[lPencils * (mx + 8)];
        break;
    case 1:
        fpDeriv[0] = derivative_y;
        fpDeriv[1] = derivative_y_lPencils;
        ispcDeriv[0] = ispc::derivative_y_ispc;
        ispcDeriv[1] = ispc::derivative_y_lPencils_ispc;
        ispc_sm_ptr[0] = new float[sPencils * (my + 8)];
        ispc_sm_ptr[1] = new float[lPencils * (my + 8)];
        break;
    case 2:
        fpDeriv[0] = derivative_z;
        fpDeriv[1] = derivative_z_lPencils;
        ispcDeriv[0] = ispc::derivative_z_ispc;
        ispcDeriv[1] = ispc::derivative_z_lPencils_ispc;
        ispc_sm_ptr[0] = new float[sPencils * (mz + 8)];
        ispc_sm_ptr[1] = new float[lPencils * (mz + 8)];
        break;
    }

    int sharedDims[3][2][2] = {mx,       sPencils, mx,       lPencils,
                               sPencils, my,       lPencils, my,
                               sPencils, mz,       lPencils, mz};

    float *f = new float[mx * my * mz];
    float *df = new float[mx * my * mz];
    float *ispc_f = new float[mx * my * mz];
    float *sol = new float[mx * my * mz];

    initInput(f, dimension);
    initSol(sol, dimension);

    // device arrays
    int bytes = mx * my * mz * sizeof(float);
    float *d_f, *d_df;
    cudaCheck(hipMalloc((void **)&d_f, bytes));
    cudaCheck(hipMalloc((void **)&d_df, bytes));

    const int nReps = 20;
    float milliseconds;
    hipEvent_t startEvent, stopEvent;
    cudaCheck(hipEventCreate(&startEvent));
    cudaCheck(hipEventCreate(&stopEvent));

    double error, maxError;

    printf("%c derivatives\n\n", (char)(0x58 + dimension));

    for (int fp = 0; fp < 2; fp++) {
        cudaCheck(hipMemcpy(d_f, f, bytes, hipMemcpyHostToDevice));
        cudaCheck(hipMemset(d_df, 0, bytes));
        memset(ispc_f, 0, bytes);

        fpDeriv[fp]<<<grid[dimension][fp], block[dimension][fp]>>>(
            d_f, d_df); // warm up
        cudaCheck(hipEventRecord(startEvent, 0));
        ispc::gridDim grid_dim{grid[dimension][fp].x, grid[dimension][fp].y,
                               grid[dimension][fp].z};
        ispc::blockDim block_dim{block[dimension][fp].x, block[dimension][fp].y,
                                 block[dimension][fp].z};
        for (int i = 0; i < nReps; i++) {
            fpDeriv[fp]<<<grid[dimension][fp], block[dimension][fp]>>>(d_f,
                                                                       d_df);
            ispcDeriv[fp](cm, ispc_sm_ptr[fp], grid_dim, block_dim, f, ispc_f,
                          pencil[fp], mx, my);
        }

        cudaCheck(hipEventRecord(stopEvent, 0));
        cudaCheck(hipEventSynchronize(stopEvent));
        cudaCheck(hipEventElapsedTime(&milliseconds, startEvent, stopEvent));

        cudaCheck(hipMemcpy(df, d_df, bytes, hipMemcpyDeviceToHost));

        checkResults(error, maxError, sol, df);

        printf("  Using shared memory tile of %d x %d\n",
               sharedDims[dimension][fp][0], sharedDims[dimension][fp][1]);
        printf("   RMS error: %e\n", error);
        printf("   MAX error: %e\n", maxError);
        printf("   Average time (ms): %f\n", milliseconds / nReps);
        printf("   Average Bandwidth (GB/s): %f\n\n",
               2.f * 1e-6 * mx * my * mz * nReps * sizeof(float) /
                   milliseconds);
        checkResults(error, maxError, sol, ispc_f);
        printf("   RMS error: %e\n", error);
        printf("   MAX error: %e\n", maxError);
    }

    cudaCheck(hipEventDestroy(startEvent));
    cudaCheck(hipEventDestroy(stopEvent));

    cudaCheck(hipFree(d_f));
    cudaCheck(hipFree(d_df));

    delete[] f;
    delete[] df;
    delete[] sol;
    delete[] ispc_f;
    delete[] ispc_sm_ptr[0];
    delete[] ispc_sm_ptr[1];
}

// This the main host code for the finite difference
// example.  The kernels are contained in the derivative_m module

int main(void) {
    // Print device and precision
    hipDeviceProp_t prop;
    cudaCheck(hipGetDeviceProperties(&prop, 0));
    printf("\nDevice Name: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n\n", prop.major, prop.minor);

    setDerivativeParameters(); // initialize

    runTest(0); // x derivative
    runTest(1); // y derivative
    // runTest(2); // z derivative

    return 0;
}